
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>


const int DSIZE = 40960;
const int block_size = 256;
const int grid_size = DSIZE/block_size;


__global__ void vector_swap(float* vec1, float* vec2, float* buffer, int size) {
    // Express the vector index in terms of threads and blocks
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Swap the vector elements - make sure you are not out of range
    if (idx < size) {
        buffer[idx] = vec1[idx];
        vec1[idx] = vec2[idx];
        vec2[idx] = buffer[idx];
    }
}

void print_vector(float* vec, int to_print=20) {
    std::cout << "[";
    for (int i = 0; i < to_print; i++) {
        std::cout << vec[i] << ", ";
    }
    std::cout << "... ]" << std::endl;
}

int main() {
    float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;
    h_A = new float[DSIZE];
    h_B = new float[DSIZE];
    h_C = new float[DSIZE];


    for (int i = 0; i < DSIZE; i++) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
        h_C[i] = 0;
    }

    print_vector(h_A);
    print_vector(h_B);

    // Allocate memory for host and device pointers 
    hipMalloc(&d_A, DSIZE * sizeof(float));
    hipMalloc(&d_B, DSIZE * sizeof(float));
    hipMalloc(&d_C, DSIZE * sizeof(float));

    // Copy from host to device
    hipMemcpy(d_A, h_A, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, DSIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, DSIZE * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    vector_swap<<<grid_size,block_size>>>(d_A, d_B, d_C, DSIZE);
    hipDeviceSynchronize();

    // Copy back to host 
    hipMemcpy(h_A, d_A, DSIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, DSIZE * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, DSIZE * sizeof(float), hipMemcpyDeviceToHost);

    // Print and check some elements to make sure swapping was successfull
    std::cout << std::endl;
    print_vector(h_A);
    print_vector(h_B);

    // Free the memory 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
